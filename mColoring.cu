
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdbool.h>
#include <sys/time.h>
#include <iostream>
#include <stdio.h>
#define LEVEL 100

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

__device__
void printSolution(int *color , int *graph, int V) {
    printf("Solution Exists:"
        " Following are the assigned colors \n");
    for (int i = 0; i < V; i++)
        printf(" %d ", color[i]);
    printf("\n");
}

/* Function to check if the color can be safely assigned */
__device__
bool isSafe (int v, int *graph, int *color, int c, int V) {
    for (int i = 0; i < V; i++)
        if (graph[v*V + i] == 1 && c == color[i])
            return false;
    return true;
}

// __device__
// void graphColoringUtil(int graph[][100], int m, int color[], int v) {
//     if(found==false) {
//         if (v == V) {
//             printSolution(color,graph);
//             return;
//         }
//         for (int c = 1; c <= m; c++) {
//         /* Check if assignment of color c to v is fine*/
//             color[v] = c;
//             if (isSafe(v, graph, color, c)) {
//                 graphColoringUtil (graph, m, color, v+1);
//             }
//         }
//         return;
//     }
// }

__global__
void graphColoringUtilParallel(int *graph, int *m, int *color, int v, bool *found, int *V, int *temp, bool *flag, hiprandState_t state, unsigned int seed) {
    if (*flag) {
        // curandState_t state;
        /* we have to initialize the state */
        hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
                0, /* the sequence number is only important with multiple cores */
                0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
                &state);
        *flag = false;
    }
    if(*found==false) {
        // for (int i = 1; i <= *m; ++i) {
        // while (1) {
            if (v == *V) {
                printSolution(color,graph, *V);
                *found = true;
                return;
            }

            color[v] = hiprand(&state)%(*m) + 1;
            // color[v] = (*temp)%(*m) + 1;

            if (isSafe(v, graph, color, color[v], *V)) {
                if (v < LEVEL) {

                    int *tempColors = new int[100];
                    // cudaMallocManaged(&color, (*V)*sizeof(int));
                    for (int j = 0; j <= v; ++j) {
                        tempColors[j] = color[j];
                    }

                    graphColoringUtilParallel<<<1,4>>>(graph, m, tempColors, v+1, found, V, temp, flag, state, seed);
                    // cudaDeviceSynchronize();
                    delete [] tempColors;
                    // #pragma omp task firstprivate(v)
                    // {
                    //     int id = omp_get_thread_num();
                    //     printf("Thread assigned %d\n",id );
                    //     graphColoringUtilParallel(graph, m, tempColors, v+1);   // generate task of serial function
                    //     graphColoringUtilParallel<<<1,8>>>(graph, m, tempColors, *v+1, found, V);
                    // }
                }
                // else{
                    // #pragma omp taskwait
                    // graphColoringUtil(graph, m, color, v+1);
                // }
            }
        // }
        for (int i = 0; i < *V; i++)
            printf("thread id %d %d ", threadIdx.x, color[i] );
        printf("\n");
        return;
    }
}

void graphColoring(int *graph, int *m, int *V, bool *found) {
    // Initialize all color values as 0.
    int *color, *temp;//, *start; // = new int[V];
    hipMallocManaged(&color, (*V)*sizeof(int));
    hipMallocManaged(&temp, sizeof(int));

    // cudaMallocManaged(&start, sizeof(int));
    // *start = 0;
    for (int i = 0; i < *V; i++)
        color[i] = 0;

    // #pragma omp parallel shared(found)
    // {
    //     #pragma omp single
    //     {
    //         graphColoringUtilParallel(graph, m, color, 0 );
    //     }
    // }
    bool *flag;
    hipMallocManaged(&flag, sizeof(bool));
    *flag = true;
    hiprandState_t state;
    graphColoringUtilParallel<<<1,1>>>(graph, m, color, 0, found, V, temp, flag, state, time(NULL));
    hipDeviceSynchronize();

    hipFree(color);
    hipFree(temp);
}


int main() {
    srand(time(NULL));

    int _vertices, _colors;
    std::cout << "Enter number of vertices: ";
    std::cin >> _vertices;
    std::cout << "Enter number of colours: ";
    std::cin >> _colors;

    struct timeval  TimeValue_Start;
    struct timezone TimeZone_Start;

    struct timeval  TimeValue_Final;
    struct timezone TimeZone_Final;
    long   time_start, time_end;
    double  time_overhead;

    // Number of vertices, colors
    int *V, *m, *graph;
    bool *found;

    hipMallocManaged(&V, sizeof(int));
    hipMallocManaged(&m, sizeof(int));
    hipMallocManaged(&graph, ((_vertices*2) + 1)*sizeof(int));
    hipMallocManaged(&found, sizeof(bool));

    *V = _vertices;
    *m = _colors;
    *found = false;

    /* Example Graph
        (3)---(2)
         |   / |
         |  /  |
         | /   |
        (0)---(1)
        {{0, 1, 1, 1},
            {1, 0, 1, 0},
            {1, 1, 0, 1},
            {1, 0, 1, 0},
        };
    */

    for(int i=0;i<_vertices;i++) {
        for (int j=0;j<_vertices;j++) {
            if(i==j)
                graph[i*_vertices + j]=0;
            else {
                graph[i*_vertices + j] = rand()%2;
                graph[j*_vertices + i] = graph[i*_vertices + j];
            }
        }
    }


    printf("Adjacency Matrix\n");

    for(int i=0;i<_vertices;i++) {
        for (int j=0;j<_vertices;j++)
            printf("%d ", graph[i*_vertices + j]);
        printf("\n");
    }

    gettimeofday(&TimeValue_Start, &TimeZone_Start);

    graphColoring (graph, m, V, found);

    if(*found==false)
        printf("No solution exists\n");

    gettimeofday(&TimeValue_Final, &TimeZone_Final);

    time_start = TimeValue_Start.tv_sec * 1000000 + TimeValue_Start.tv_usec;
    time_end = TimeValue_Final.tv_sec * 1000000 + TimeValue_Final.tv_usec;
    time_overhead = (time_end - time_start)/1000000.0;

    printf("\n Time in Seconds (T)  : %lf",time_overhead);

    hipFree(V);
    hipFree(m);
    hipFree(found);
    hipFree(graph);

    return 0;
}
